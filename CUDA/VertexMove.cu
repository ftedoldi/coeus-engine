#include "hip/hip_runtime.h"
#include "VertexMove.cuh"
#include "../Includes/CUDA/hip/hip_runtime.h"
#include "../Includes/CUDA/"

#include <stdio.h>

__global__ void kernelMoveVertices(float3* dptr, size_t numVertices, double dt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < numVertices)
        dptr[tid].x += dt;
    
}

void MoveVertices(hipGraphicsResource_t& vbo, size_t numVertices, double dt)
{
    dim3 block(16, 1, 1);
    dim3 grid((numVertices + block.x - 1) / block.x, 1, 1);
    float3* dptr;
    size_t vs_dst;

    // Map the resources so they can be used in the kernel.
    hipGraphicsMapResources(1, &vbo);
    
    hipGraphicsResourceGetMappedPointer((void**)&dptr, &vs_dst, vbo);

    //Kernel call
    kernelMoveVertices<<<block, grid>>>(dptr, numVertices, dt);

    hipGraphicsUnmapResources(1, &vbo);
}