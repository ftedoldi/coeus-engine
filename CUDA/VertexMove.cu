#include "hip/hip_runtime.h"
#include "VertexMove.cuh"
#include "../Includes/CUDA/hip/hip_runtime.h"
#include "../Includes/CUDA/"

#include <stdio.h>

__global__ void kernelMoveVertices(float3* dptr, size_t numVertices)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    dptr[tid].x += 1.0f;
    
}

void MoveVertices(hipGraphicsResource_t& vbo, size_t numVertices)
{
    dim3 block(8, 1, 1);
    dim3 grid(numVertices/block.x, 1, 1);
    float3* dptr;
    size_t vs_dst;

    // Map the resources so they can be used in the kernel.
    hipGraphicsMapResources(1, &vbo);
    
    hipGraphicsResourceGetMappedPointer((void**)&dptr, &vs_dst, vbo);

    //Kernel call
    kernelMoveVertices<<<block, grid>>>(dptr, numVertices);

    hipGraphicsUnmapResources(1, &vbo);
}