#include "hip/hip_runtime.h"
#include "VectorOperations.cuh"

#include "../Includes/CUDA/hip/hip_runtime.h"
#include "../Includes/CUDA/"

#include <stdio.h>

__global__ void Vector3AddKernel(float* a, float* b, float* c, int sizeOfArrays)
{
    int i = threadIdx.x;

    if (i < sizeOfArrays)
        c[i] = a[i] + b[i];
}

__global__ void Matrix3ToMatrix4Kernel(float* m3, float* m4)
{
    int x = threadIdx.x;
    int y = threadIdx.y;

    int flatIndex = x + (4 * y);

    if ( x < 3 && y < 3 )
        m4[flatIndex] = m3[x + 3 * y];
    else
        m4[flatIndex] = 0;
}

int AddVectors3(float vector1[3], float vector2[3], float result[3])
{
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, 3 * sizeof(float));
    hipMalloc(&d_b, 3 * sizeof(float));
    hipMalloc(&d_c, 3 * sizeof(float));

    hipMemcpy(d_a, vector1, 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, vector2, 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, result, 3 * sizeof(float), hipMemcpyHostToDevice);

    Vector3AddKernel<<<1, 3>>>(d_a, d_b, d_c, 3);

    hipMemcpy(result, d_c, 3 * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 1;
}

int Matrix3ToMatrix4(float m3[9], float m4[16])
{
    float *d_m3, *d_m4;

    hipMalloc(&d_m3, 3 * 3 * sizeof(float));
    hipMalloc(&d_m4, 4 * 4 * sizeof(float));

    hipMemcpy(d_m3, m3, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_m4, m4, 4 * 4 * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid (1);
    dim3 block (4, 4);

    Matrix3ToMatrix4Kernel<<<grid, block>>>(d_m3, d_m4);

    hipMemcpy(m4, d_m4, 4 * 4 * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    
    hipFree(d_m3);
    hipFree(d_m4);

    m4[15] = 1;

    return 1;
}